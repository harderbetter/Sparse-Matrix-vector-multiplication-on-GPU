
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h> 
#include <sys/time.h>

__global__ void mv(int num_rows,int *ptr,int *indices,float *data,float *x,float *y)
{   int jj;
    int row= threadIdx.x + blockIdx.x*blockDim.x;
	if(row< num_rows){
		float dot=0;
		
		int row_start = ptr[row];
		int row_end = ptr[row+1];
		for ( jj = row_start; jj<row_end; jj++)
			dot += data[jj] * x[indices[jj]];
		y[row] = dot;
		
	}
}
int main()
{   
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop); 
	int num_rows=1000;
	int ptr[1001];
	int indices[25000];
	float data[25000];
	int i,j,m,n,q;
	for( i=0; i<1001; i++)
	{
		ptr[i]=25*i;
	}

	
	for( j=0; j<25000;j++)
	{
		indices[j]=j;
	}
	
	for(m=0;m<1000;m++)
	{
		for(n=0;n<25;n++)
		{
			data[25*m+n]=n;
		}
	}
	
 
	float v[25000];
	
	for(q=0;q<25000;q++)
	{
		v[q]=q;
	}
 
	
	
	float *outcome;
	outcome= (float*)malloc(1000*sizeof(float));
	memset(outcome,0,1000*sizeof(float));
		int *p,*in;
	float *da,*x,*y;
 
	hipMalloc(&p,1001*sizeof(int));
	hipMemcpy(p,ptr,1001*sizeof(int),hipMemcpyHostToDevice);
	hipMalloc(&in,25000*sizeof(int));
	hipMemcpy(in,indices,25000*sizeof(int),hipMemcpyHostToDevice);
	hipMalloc(&da,25000*sizeof(float));
	hipMemcpy(da,data,25000*sizeof(float),hipMemcpyHostToDevice);
	hipMalloc(&x,25000*sizeof(float));
	hipMemcpy(x,v,25000*sizeof(float),hipMemcpyHostToDevice);
	hipMalloc(&y,1000*sizeof(float));
 
 
	dim3 blocks(49);
	dim3 threads(25000);
	hipEventRecord(start);	
	mv<<<blocks,threads>>>(num_rows,p,in,da,x,y);
	hipEventRecord(stop);
	hipMemcpy(outcome,y,sizeof(float)*1000,hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("calculate time on gpu (ms): %f\n", milliseconds);
	
 
	free(outcome);
	hipFree(p);
	hipFree(in);
	hipFree(da);
	hipFree(x);
	hipFree(y);
	hipDeviceReset();
	return EXIT_SUCCESS;
	
	

}
