#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h> 
#include <sys/time.h>


__global__ void mv(int num_rows,int *ptr,int *indices,float *data,float *x,float *y)
{   __shared__ float vals[128];
    int thread_id = threadIdx.x + blockIdx.x*blockDim.x;   // global thread index
    int warp_id = thread_id / 32;                         // global warp index
	int lane = thread_id & (32-1);                         // thread index within the warp
	// one warp per row
	int row = warp_id;
    int jj;
	if(row< num_rows){
		
		int row_start = ptr[row];
		int row_end = ptr[row+1];
		
		//compute running sum per thread
		vals[threadIdx.x] = 0;
		for ( jj = row_start + lane; jj<row_end; jj+=32)
			vals[threadIdx.x] += data[jj] * x[indices[jj]];
		//parallel reduction in shared memory
		if(lane < 16) vals[threadIdx.x] += vals[threadIdx.x +16];
		if(lane < 8) vals[threadIdx.x] += vals[threadIdx.x +8];
		if(lane < 4) vals[threadIdx.x] += vals[threadIdx.x +4];
		if(lane < 2) vals[threadIdx.x] += vals[threadIdx.x +2];
		if(lane < 1) vals[threadIdx.x] += vals[threadIdx.x +1];
		
		// first thread writes the result
		if(lane == 0)
		y[row] = vals[threadIdx.x];
		
	}
}
int main()
{   
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop); 
	int num_rows=1000;
	int ptr[1001];
	int indices[25000];
	float data[25000];
	int i,j,m,n,q;
	for( i=0; i<1001; i++)
	{
		ptr[i]=25*i;
	}

	
	for( j=0; j<25000;j++)
	{
		indices[j]=j;
	}
	
	for(m=0;m<1000;m++)
	{
		for(n=0;n<25;n++)
		{
			data[25*m+n]=n;
		}
	}
	
 
	float v[25000];
	
	for(q=0;q<25000;q++)
	{
		v[q]=q;
	}
 
	
	
	float *outcome;
	outcome= (float*)malloc(1000*sizeof(float));
	memset(outcome,0,1000*sizeof(float));
		int *p,*in;
	float *da,*x,*y;
 
	hipMalloc(&p,1001*sizeof(int));
	hipMemcpy(p,ptr,1001*sizeof(int),hipMemcpyHostToDevice);
	hipMalloc(&in,25000*sizeof(int));
	hipMemcpy(in,indices,25000*sizeof(int),hipMemcpyHostToDevice);
	hipMalloc(&da,25000*sizeof(float));
	hipMemcpy(da,data,25000*sizeof(float),hipMemcpyHostToDevice);
	hipMalloc(&x,25000*sizeof(float));
	hipMemcpy(x,v,25000*sizeof(float),hipMemcpyHostToDevice);
	hipMalloc(&y,1000*sizeof(float));
 
 
	dim3 blocks(63);
	dim3 threads(32000);
	hipEventRecord(start);	
	mv<<<blocks,threads>>>(num_rows,p,in,da,x,y);
	hipEventRecord(stop);
	hipMemcpy(outcome,y,sizeof(float)*1000,hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("calculate time on gpu (ms): %f\n", milliseconds);
	
 
	free(outcome);
	hipFree(p);
	hipFree(in);
	hipFree(da);
	hipFree(x);
	hipFree(y);
	hipDeviceReset();
	return EXIT_SUCCESS;
	
	

}