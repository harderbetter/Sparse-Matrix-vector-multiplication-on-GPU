#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h> 
#include <sys/time.h>


__global__ void mv(int num_rows,int *ptr,int *indices,float *data,float *x,float *y)
{   __shared__ float vals[128];
    int thread_id = threadIdx.x + blockIdx.x*blockDim.x;   // global thread index
    int warp_id = thread_id / 32;                         // global warp index
	int lane = thread_id & (32-1);                         // thread index within the warp
	// one warp per row
	int row = warp_id;
    int jj;
	if(row< num_rows){
		
		int row_start = ptr[row];
		int row_end = ptr[row+1];
		
		//compute running sum per thread
		vals[threadIdx.x] = 0;
		for ( jj = row_start + lane; jj<row_end; jj+=32)
			vals[threadIdx.x] += data[jj] * x[indices[jj]];
		//parallel reduction in shared memory
		if(lane < 16) vals[threadIdx.x] += vals[threadIdx.x +16];
		if(lane < 8) vals[threadIdx.x] += vals[threadIdx.x +8];
		if(lane < 4) vals[threadIdx.x] += vals[threadIdx.x +4];
		if(lane < 2) vals[threadIdx.x] += vals[threadIdx.x +2];
		if(lane < 1) vals[threadIdx.x] += vals[threadIdx.x +1];
		
		// first thread writes the result
		if(lane == 0)
		y[row] = vals[threadIdx.x];
		
	}
}
int main()
{   
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop); 
	int num_rows=4;
	int ptr[] = {0,2,4,7,9};
	int indices[] = {1,2,0,2,0,2,3,1,3};
	float data[] = {2,6,1,7,5,3,9,5,3};
 
	float v[]={1,2,3,4};
 
	
 
	float *outcome;
	outcome= (float*)malloc(4*sizeof(float));
	memset(outcome,0,4*sizeof(float));
 

	int *p,*in;
	float *da,*x,*y;
 
	hipMalloc(&p,5*sizeof(int));
	hipMemcpy(p,ptr,5*sizeof(int),hipMemcpyHostToDevice);
	hipMalloc(&in,9*sizeof(int));
	hipMemcpy(in,indices,9*sizeof(int),hipMemcpyHostToDevice);
	hipMalloc(&da,9*sizeof(float));
	hipMemcpy(da,data,9*sizeof(float),hipMemcpyHostToDevice);
	hipMalloc(&x,4*sizeof(float));
	hipMemcpy(x,v,4*sizeof(float),hipMemcpyHostToDevice);
	hipMalloc(&y,4*sizeof(float));
 
 
	dim3 blocks(1);
	dim3 threads(4*32);
	hipEventRecord(start);	
	mv<<<blocks,threads>>>(num_rows,p,in,da,x,y);
	hipEventRecord(stop);
	hipMemcpy(outcome,y,sizeof(float)*4,hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("calculate time on gpu (ms): %f\n", milliseconds);
	printf("the outcome is \n");
	for(int i=0;i< num_rows; i++)
	{
		printf("%f \n",outcome[i]);
	}
 
	free(outcome);
	hipFree(p);
	hipFree(in);
	hipFree(da);
	hipFree(x);
	hipFree(y);
	hipDeviceReset();
	return EXIT_SUCCESS;
 
}