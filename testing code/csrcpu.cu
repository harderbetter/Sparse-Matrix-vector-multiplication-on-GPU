#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h> 
#include <sys/time.h>
void cpu(int num_rows,int *ptr, int *indices,float *data,float *x,float *y)
{ int i,jj;
 for( i=0;i<4;i++)
 {
	if(i< num_rows)
	{
		float dot=0;
		
		int row_start = ptr[i];
		int row_end = ptr[i+1];
		for (jj = row_start; jj<row_end; jj++)
			dot += data[jj] * x[indices[jj]];
		y[i] = dot;
		
	}
 }
}
int main()
{
	int num_rows=4;
	int ptr[] = {0,2,4,7,9};
	int indices[] = {1,2,0,2,0,2,3,1,3};
	float data[] = {2,6,1,7,5,3,9,5,3};
 
	float v[]={1,2,3,4};
 
	float *z;
	z= (float*)malloc(4*sizeof(float));
	memset(z,0,4*sizeof(float));

	
	cpu(num_rows,ptr,indices,data,v,z);
	
	
	
	printf("the outcome in cpu is \n");
	for(int i=0;i< num_rows; i++)
	{
		printf("%f \n",z[i]);
	}
}
