#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h> 
#include <sys/time.h>


__global__ void mv(int num_rows,int *ptr,int *indices,float *data,float *x,float *y)
{   int jj;
    int row= threadIdx.x + blockIdx.x*blockDim.x;
	if(row< num_rows){
		float dot=0;
		
		int row_start = ptr[row];
		int row_end = ptr[row+1];
		for ( jj = row_start; jj<row_end; jj++)
			dot += data[jj] * x[indices[jj]];
		y[row] = dot;
		
	}
}
int main()
{   
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop); 
	int num_rows=4;
	int ptr[] = {0,2,4,7,9};
	int indices[] = {1,2,0,2,0,2,3,1,3};
	float data[] = {2,6,1,7,5,3,9,5,3};
 
	float v[]={1,2,3,4};
 
	
 
	float *outcome;
	outcome= (float*)malloc(4*sizeof(float));
	memset(outcome,0,4*sizeof(float));
 

	int *p,*in;
	float *da,*x,*y;
 
	hipMalloc(&p,5*sizeof(int));
	hipMemcpy(p,ptr,5*sizeof(int),hipMemcpyHostToDevice);
	hipMalloc(&in,9*sizeof(int));
	hipMemcpy(in,indices,9*sizeof(int),hipMemcpyHostToDevice);
	hipMalloc(&da,9*sizeof(float));
	hipMemcpy(da,data,9*sizeof(float),hipMemcpyHostToDevice);
	hipMalloc(&x,4*sizeof(float));
	hipMemcpy(x,v,4*sizeof(float),hipMemcpyHostToDevice);
	hipMalloc(&y,4*sizeof(float));
 
 
	dim3 blocks(1);
	dim3 threads(4);
	hipEventRecord(start);	
	mv<<<blocks,threads>>>(num_rows,p,in,da,x,y);
	hipEventRecord(stop);
	hipMemcpy(outcome,y,sizeof(float)*4,hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("calculate time on gpu (ms): %f\n", milliseconds);
	printf("the outcome is \n");
	for(int i=0;i< num_rows; i++)
	{
		printf("%f \n",outcome[i]);
	}
 
	free(outcome);
	hipFree(p);
	hipFree(in);
	hipFree(da);
	hipFree(x);
	hipFree(y);
	hipDeviceReset();
	return EXIT_SUCCESS;
 
}